#include "hip/hip_runtime.h"
#include </usr/local/cuda/include/hip/hip_runtime.h>
#include </usr/local/cuda/include/hiprand.h>
#include </usr/local/cuda/include/hiprand/hiprand_kernel.h>
#include "../include/ptrKernel.h"
#include <cstdlib>
#include <iostream>


__global__ void configureInitialConditions(size_t size, int *statesPtr, int *neighborsPtr, int *evolutionPtr, double densityStatesAB, double densityStatesAC, float *randomNumbers){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < size) {
        float randVal1{randomNumbers[tid]}, randVal2{randomNumbers[tid + size]};
        // Create a variable that will determines if the state is or not zero
        int selectZero{1- int(randVal1 + densityStatesAB)};

        // Initialize states randomly considering states density
        if(selectZero == 0){statesPtr[tid] = 0;}
        else{statesPtr[tid] = 1 - 2 * int(randVal2 + densityStatesAC);}

        // As initial condition we impose that neighbors are equal to states for the first step
        neighborsPtr[tid] = statesPtr[tid];

        // Finally replace the garbage on evolution buffer
        evolutionPtr[tid] = 5;

        // Check for evelements out of range
        tid += blockDim.x * gridDim.x;
    }
}

// Display pointer
void displayPtr(size_t ptrSize, int *ptr){
    std::cout << "[" << ptr[0] << ", ";
    for(size_t i{}; i < ptrSize - 1; i++){
        std::cout << ptr[i] << ", ";
    }
    std::cout << ptr[ptrSize-1] << "]" << std::endl;
}
