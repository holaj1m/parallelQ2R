#include <iostream>
#include <cstdlib>

#include </usr/local/cuda/include/hip/hip_runtime.h>
#include </usr/local/cuda/include/hiprand.h>
#include </usr/local/cuda/include/hiprand/hiprand_kernel.h>

#include "../include/errorHandling.h"
#include "../include/ptrKernel.h"
#include "../include/evolutionRuleKernel.h"


int main(){

    // Size of the system
    size_t dimension{10};

    // Pointers on host to allocate states of the system
    int *currentStates{nullptr}, *neighbors{nullptr}, *nextStates{nullptr};
    
    // Pointers on host to al

    // Allocate memory on the CPU
    currentStates   = new int[dimension];
    neighbors       = new int[dimension];
    nextStates      = new int[dimension];

    //============================================================================
    //============================ D E V I C E ===================================
    // Pointers to handle device computations 
    int *d_currentStates{nullptr}, *d_neighbors{nullptr}, *d_nextStates{nullptr};

    // Allocate memory on the GPU to handle states
    HANDLE_ERROR(hipMalloc((void**)&d_currentStates, dimension * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&d_neighbors, dimension * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&d_nextStates, dimension * sizeof(int)));

    //-------------------- R A N D O M  N U M B E R S ------------------------------
    // Allocate memory on the GPU to generate initial condition
    float *d_randNumbers{nullptr};
    // Consider dimension random numbers
    HANDLE_ERROR(hipMalloc((void**)&d_randNumbers, 2 * dimension * sizeof(float)));

    hiprandGenerator_t gen;
    // Create ther random number generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW));
    // Set the seed
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL));
    // Generate dimension random numbers
    CURAND_CALL(hiprandGenerateUniform(gen, d_randNumbers, 2 * dimension));
    // Destroy the generator
    CURAND_CALL(hiprandDestroyGenerator(gen));
    //------------------------------------------------------------------------------

    // Density between states A and B, and between A and C.
    double densityStatesAB{0.3};
    double densityStatesAC{0.3};

    // Call the kernel to configure initial conditions
    configureInitialConditions<<<1,10>>>(dimension, d_currentStates, d_neighbors, d_nextStates, densityStatesAB, densityStatesAC, d_randNumbers);

    

    // ---------------------- E V O L V E  T H E  S Y S T E M --------------------------
    //----------------------------------------------------------------------------------
    size_t time{10};
    for(size_t t{}; t < time; t++){
        // Apply the rule
        Q2RPottsRule<<<1,10>>>(dimension, d_currentStates, d_neighbors, d_nextStates);

        // Ensure kernel is complete before memcpy
        HANDLE_ERROR(hipDeviceSynchronize()); 

        // VERIFICATION
        // Copy the results to the CPU
        HANDLE_ERROR(hipMemcpy(currentStates, d_currentStates, dimension * sizeof(int), hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(neighbors, d_neighbors, dimension * sizeof(int), hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(nextStates, d_nextStates, dimension * sizeof(int), hipMemcpyDeviceToHost));

        std::cout << "======================================================" << std::endl;
        std::cout << "============== T I M E  "<< t <<"  ===================" << std::endl;
        std::cout << "The elements stored on current states are: " << std::endl;
        displayPtr(dimension,currentStates);
        std::cout << "The elements stored on neighbors are: " << std::endl;
        displayPtr(dimension,neighbors);
        std::cout << "The elements stored on next states are: " << std::endl;
        displayPtr(dimension,nextStates); 

        // Transfer values from neighbors to current states
        HANDLE_ERROR(hipMemcpy(d_currentStates, d_neighbors, dimension * sizeof(int), hipMemcpyDeviceToDevice));

        // Transfer the values from next states to neighbors
        HANDLE_ERROR(hipMemcpy(d_neighbors, d_nextStates, dimension * sizeof(int), hipMemcpyDeviceToDevice));
    }


    // Copy the results to the CPU
    HANDLE_ERROR(hipMemcpy(currentStates, d_currentStates, dimension * sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(neighbors, d_neighbors, dimension * sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(nextStates, d_nextStates, dimension * sizeof(int), hipMemcpyDeviceToHost));
    //============================================================================
    //============================== H O S T =====================================
    // Verify elements on pointers
    /*std::cout << "The elements stored on current states are: " << std::endl;
    displayPtr(50,currentStates);
    std::cout << "The elements stored on neighbors are: " << std::endl;
    displayPtr(50,neighbors);
    std::cout << "The elements stored on next states are: " << std::endl;
    displayPtr(50,nextStates);*/

    //============================================================================
    //============================ D E V I C E ===================================
    // Clean te allocated memory on GPU
    HANDLE_ERROR(hipFree(d_currentStates));
    HANDLE_ERROR(hipFree(d_neighbors));
    HANDLE_ERROR(hipFree(d_nextStates));

    HANDLE_ERROR(hipFree(d_randNumbers));
    
    //============================================================================

    // Clean the allocated memory on GPU
    delete[] currentStates; currentStates = nullptr;
    delete[] neighbors; neighbors = nullptr;
    delete[] nextStates; nextStates = nullptr;

    return 0;
}