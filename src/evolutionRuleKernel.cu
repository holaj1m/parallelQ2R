#include "hip/hip_runtime.h"
#include </usr/local/cuda/include/hip/hip_runtime.h>
#include <cstdlib>


#include "../include/evolutionRuleKernel.h"

__global__ void Q2RPottsRule(size_t size, int *statesPtr, int *neighborsPtr, int *evolutionPtr){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < size) {
        // Mod operation to obtain neighbors
        size_t firstNeighborRightIdx  =   (tid + 1) % size;
        size_t secondNeighborRightIdx =   (tid + 2) % size;

        size_t firstNeighborLeftIdx   =   (tid + size - 1) % size;
        size_t secondNeighborLeftIdx  =   (tid + size - 2) % size;

        // Initialize the current state and its neighbors
        int currentState{statesPtr[tid]};
        int firstNeighborRight{neighborsPtr[firstNeighborRightIdx]};
        int secondNeighborRight{neighborsPtr[secondNeighborRightIdx]};
        int firstNeighborLeft{neighborsPtr[firstNeighborLeftIdx]};
        int secondNeighborLeft{neighborsPtr[secondNeighborLeftIdx]};

        // Create an array to store the neighborhood
        int neighborhood[4] = {firstNeighborRight, secondNeighborRight, firstNeighborLeft, secondNeighborLeft};

        // Create variables to count the frequency of each state on neighborhood 
        int freqStateA{}, freqStateB{}, freqStateC{};

        //Count the frequency
        for(size_t element; element < 4; element++){
            switch (neighborhood[element])
            {
            case -1: freqStateA++; break;
            case 0: freqStateB++; break;
            default: freqStateC++;
            }
        }

        // Update the evolution state considering the different frequency among neighborhood
        if(freqStateA == 4 || freqStateB == 4 || freqStateC == 4){
        evolutionPtr[tid] = currentState;
        }

        else if(freqStateA < 3 && freqStateB < 3 && freqStateC < 3){

            if(freqStateA == 0 || freqStateB == 0 || freqStateC == 0){

                if(freqStateA == 2 && freqStateB == 2){
                    switch(currentState){
                        case -1: evolutionPtr[tid] = 0; break;
                        case 0: evolutionPtr[tid] = -1; break;
                        default: evolutionPtr[tid] = 1; break;
                    }
                }

                else if(freqStateA == 2 && freqStateC == 2){
                    switch(currentState){
                        case -1: evolutionPtr[tid] = 1; break;
                        case 1: evolutionPtr[tid] = -1; break;
                        default: evolutionPtr[tid] = 0; 
                    }
                }

                else if(freqStateB == 2 && freqStateC == 2){
                    switch(currentState){
                        case 0: evolutionPtr[tid] = 1; break;
                        case 1: evolutionPtr[tid] = 0; break;
                        default: evolutionPtr[tid] = -1;
                    }
                }

            }

            else{
                if(freqStateA == 2){
                    switch(currentState){
                        case -1: evolutionPtr[tid] = -1; break;
                        case 0: evolutionPtr[tid] = 1; break;
                        default: evolutionPtr[tid] = 0;
                    }
                }

                else if(freqStateB == 2){
                    switch(currentState){
                        case 0: evolutionPtr[tid] = 0; break;
                        case 1: evolutionPtr[tid] = -1; break;
                        default: evolutionPtr[tid] = 1; 
                    }
                }

                else if(freqStateC == 2){
                    switch(currentState){
                        case 1: evolutionPtr[tid] = 1; break;
                        case -1: evolutionPtr[tid] = 0; break;
                        default: evolutionPtr[tid] = -1;
                    }
                }

            }

        }
        else{evolutionPtr[tid] = currentState;}

        // Check for elements out of range
        tid += blockDim.x * gridDim.x;
    }
}